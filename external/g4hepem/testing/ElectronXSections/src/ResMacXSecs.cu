#include "hip/hip_runtime.h"

#include "Declaration.hh"

#include "G4HepEmData.hh"
#include "G4HepEmElectronData.hh"

// don't worry it's just for testing
#define private public
#include "G4HepEmElectronManager.hh"

#include <hip/hip_runtime.h>
#include "G4HepEmCuUtils.hh"

// Pull in implementation
#include "G4HepEmElectronManager.icc"
#include "G4HepEmRunUtils.icc"

 __global__
 void TestResMacXSecDataKernel ( const struct G4HepEmElectronData* theElectronData_d,
                                 int* tsInImc_d, double* tsInEkin_d, double* tsInLogEkin_d,
                                 double* tsOutRes_d, bool isIoni, int numTestCases) {
   for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numTestCases; i += blockDim.x * gridDim.x) {
     G4HepEmElectronManager theElectronMgr;
     tsOutRes_d[i] = theElectronMgr.GetRestMacXSec (theElectronData_d, tsInImc_d[i], tsInEkin_d[i], tsInLogEkin_d[i], isIoni);
   }
 }

void TestResMacXSecDataOnDevice ( const struct G4HepEmData* hepEmData, int* tsInImc_h,
     double* tsInEkinIoni_h, double* tsInLogEkinIoni_h, double* tsInEkinBrem_h, double* tsInLogEkinBrem_h,
     double* tsOutResMXIoni_h, double* tsOutResMXBrem_h, int numTestCases, bool iselectron ) {
  //
  // --- Allocate device side memory for the input/output data and copy all input
  //     data from host to device
  int*             tsInImc_d = nullptr;
  double*     tsInEkinIoni_d = nullptr;
  double*  tsInLogEkinIoni_d = nullptr;
  double*     tsInEkinBrem_d = nullptr;
  double*  tsInLogEkinBrem_d = nullptr;
  double*   tsOutResMXIoni_d = nullptr;
  double*   tsOutResMXBrem_d = nullptr;
  //
  gpuErrchk ( hipMalloc ( &tsInImc_d,         sizeof( int )    * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsInEkinIoni_d,    sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsInLogEkinIoni_d, sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsInEkinBrem_d,    sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsInLogEkinBrem_d, sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsOutResMXIoni_d,  sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsOutResMXBrem_d,  sizeof( double ) * numTestCases ) );
  //
  // --- Copy the input data from host to device (test material-cut index, ekin and log-ekin arrays)
  gpuErrchk ( hipMemcpy ( tsInImc_d,         tsInImc_h,         sizeof( int )    * numTestCases, hipMemcpyHostToDevice) );
  gpuErrchk ( hipMemcpy ( tsInEkinIoni_d,    tsInEkinIoni_h,    sizeof( double ) * numTestCases, hipMemcpyHostToDevice) );
  gpuErrchk ( hipMemcpy ( tsInLogEkinIoni_d, tsInLogEkinIoni_h, sizeof( double ) * numTestCases, hipMemcpyHostToDevice) );
  gpuErrchk ( hipMemcpy ( tsInEkinBrem_d,    tsInEkinBrem_h,    sizeof( double ) * numTestCases, hipMemcpyHostToDevice) );
  gpuErrchk ( hipMemcpy ( tsInLogEkinBrem_d, tsInLogEkinBrem_h, sizeof( double ) * numTestCases, hipMemcpyHostToDevice) );
  //
  // --- Launch the kernels
  int numThreads = 512;
  int numBlocks  = std::ceil( float(numTestCases)/numThreads );
  //  std::cout << " N = " << numTestCases << " numBlocks = " << numBlocks << " numThreads = " << numThreads << " x = " << numBlocks*numThreads << std::endl;
  const G4HepEmElectronData* theElectronData_d = iselectron ? hepEmData->fTheElectronData_gpu : hepEmData->fThePositronData_gpu;
  // ioni
  TestResMacXSecDataKernel <<< numBlocks, numThreads >>> (theElectronData_d, tsInImc_d, tsInEkinIoni_d, tsInLogEkinIoni_d, tsOutResMXIoni_d, true,  numTestCases );
  // brem
  TestResMacXSecDataKernel <<< numBlocks, numThreads >>> (theElectronData_d, tsInImc_d, tsInEkinBrem_d, tsInLogEkinBrem_d, tsOutResMXBrem_d, false, numTestCases );
  //
  // --- Synchronize to make sure that completed on the device
  hipDeviceSynchronize();
  //
  // --- Copy the results from the device to the host
  gpuErrchk ( hipMemcpy ( tsOutResMXIoni_h,     tsOutResMXIoni_d,     sizeof( double ) * numTestCases, hipMemcpyDeviceToHost ) );
  gpuErrchk ( hipMemcpy ( tsOutResMXBrem_h,     tsOutResMXBrem_d,     sizeof( double ) * numTestCases, hipMemcpyDeviceToHost ) );
  //
  // --- Free all dynamically allocated (device side) memory
  hipFree ( tsInImc_d          );
  hipFree ( tsInEkinIoni_d    );
  hipFree ( tsInLogEkinIoni_d );
  hipFree ( tsInEkinBrem_d    );
  hipFree ( tsInLogEkinBrem_d );
  hipFree ( tsOutResMXIoni_d  );
  hipFree ( tsOutResMXBrem_d  );
}
