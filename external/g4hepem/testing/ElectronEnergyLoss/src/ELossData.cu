#include "hip/hip_runtime.h"
#include "Declaration.hh"

#include "G4HepEmData.hh"
#include "G4HepEmElectronData.hh"

// don't worry it's just for testing
#define private public
#include "G4HepEmElectronManager.hh"

#include <hip/hip_runtime.h>
#include "G4HepEmCuUtils.hh"

// Pull in implementation
#include "G4HepEmElectronManager.icc"
#include "G4HepEmRunUtils.icc"

__global__
void TestElossDataKernel  ( struct G4HepEmElectronData* theElectronData_d, int* tsInImc_d,
                            double* tsInEkin_d, double* tsInLogEkin_d, double* tsOutResRange_d,
                            double* tsOutResDEDX_d, double* tsOutResInvRange_d, int numTestCases ) {
   for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numTestCases; i += blockDim.x * gridDim.x) {
     G4HepEmElectronManager theElectronMgr;
     tsOutResRange_d[i]    = theElectronMgr.GetRestRange(theElectronData_d, tsInImc_d[i], tsInEkin_d[i], tsInLogEkin_d[i]);
     tsOutResDEDX_d[i]     = theElectronMgr.GetRestDEDX (theElectronData_d, tsInImc_d[i], tsInEkin_d[i], tsInLogEkin_d[i]);
     tsOutResInvRange_d[i] = theElectronMgr.GetInvRange (theElectronData_d, tsInImc_d[i], tsOutResRange_d[i]);
   }
 }

void TestElossDataOnDevice ( const struct G4HepEmData* hepEmData,
     int* tsInImc_h, double* tsInEkin_h, double* tsInLogEkin_h,
     double* tsOutResRange_h, double* tsOutResDEDX_h, double* tsOutResInvRange_h,
     int numTestCases, bool iselectron ) {
  //
  // --- Allocate device side memory for the input/output data and copy all input
  //     data from host to device
  int*             tsInImc_d = nullptr;
  double*         tsInEkin_d = nullptr;
  double*      tsInLogEkin_d = nullptr;
  double*     tsOutResDEDX_d = nullptr;
  double*    tsOutResRange_d = nullptr;
  double* tsOutResInvRange_d = nullptr;
  //
  gpuErrchk ( hipMalloc ( &tsInImc_d,          sizeof( int )    * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsInEkin_d,         sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsInLogEkin_d,      sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsOutResDEDX_d,     sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsOutResRange_d,    sizeof( double ) * numTestCases ) );
  gpuErrchk ( hipMalloc ( &tsOutResInvRange_d, sizeof( double ) * numTestCases ) );
  //
  // --- Copy the input data from host to device (test material-cut index, ekin and log-ekin arrays)
  gpuErrchk ( hipMemcpy ( tsInImc_d,     tsInImc_h,     sizeof( int )    * numTestCases, hipMemcpyHostToDevice) );
  gpuErrchk ( hipMemcpy ( tsInEkin_d,    tsInEkin_h,    sizeof( double ) * numTestCases, hipMemcpyHostToDevice) );
  gpuErrchk ( hipMemcpy ( tsInLogEkin_d, tsInLogEkin_h, sizeof( double ) * numTestCases, hipMemcpyHostToDevice) );
  //
  // --- Launch the kernels
  int numThreads = 512;
  int numBlocks  = std::ceil( float(numTestCases)/numThreads );
  // std::cout << " N = " << numTestCases << " numBlocks = " << numBlocks << " numThreads = " << numThreads << " x = " << numBlocks*numThreads << std::endl;
  struct G4HepEmElectronData* elData_d = iselectron ? hepEmData->fTheElectronData_gpu : hepEmData->fThePositronData_gpu;
  TestElossDataKernel <<< numBlocks, numThreads >>> (elData_d, tsInImc_d, tsInEkin_d, tsInLogEkin_d, tsOutResRange_d, tsOutResDEDX_d, tsOutResInvRange_d, numTestCases );
  //
  // --- Synchronize to make sure that completed on the device
  hipDeviceSynchronize();
  //
  // --- Copy the results from the device to the host
  gpuErrchk ( hipMemcpy ( tsOutResDEDX_h,     tsOutResDEDX_d,     sizeof( double ) * numTestCases, hipMemcpyDeviceToHost ) );
  gpuErrchk ( hipMemcpy ( tsOutResRange_h,    tsOutResRange_d,    sizeof( double ) * numTestCases, hipMemcpyDeviceToHost ) );
  gpuErrchk ( hipMemcpy ( tsOutResInvRange_h, tsOutResInvRange_d, sizeof( double ) * numTestCases, hipMemcpyDeviceToHost ) );
  //
  // --- Free all dynamically allocated (device side) memory
  hipFree ( tsInImc_d          );
  hipFree ( tsInEkin_d         );
  hipFree ( tsInLogEkin_d      );
  hipFree ( tsOutResDEDX_d     );
  hipFree ( tsOutResRange_d    );
  hipFree ( tsOutResInvRange_d );
}
